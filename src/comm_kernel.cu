#include "hip/hip_runtime.h"
/*******************************************************************************
 ******************************** BLUEFISH-1.0 *********************************
 *******************************************************************************
 *
 *  Copyright 2012 - 2014 Adam Sierakowski, The Johns Hopkins University
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *  Please contact the Johns Hopkins University to use Bluefish for
 *  commercial and/or for-profit applications.
 ******************************************************************************/

#include "cuda_comm.h"

__global__ void touchp(real *p, dom_struct *dom, int dev)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  if(tj <= dom->Gcc._jeb && tk <= dom->Gcc._keb) {
    for(int i = dom->Gcc._isb; i <= dom->Gcc._ieb; i++) {
      p[i + tj*dom->Gcc.s1b + tk*dom->Gcc.s2b] = (real)dev;
    }
  }
}
